// nvcc -arch=sm_20 10.cu
// Find the sum of all primes below 2 million

#include <stdio.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK	512
#define START_NUMBER		1414
#define TOTAL_THREADS		((2000000-START_NUMBER)/2)

__shared__ int blockPrimes[THREADS_PER_BLOCK];

// Kernel that executes on the CUDA device
__global__ void sum_primes(int *firstPrimes, size_t n, unsigned long long *blockSums)
{
	int i;
	int idx;
	int num;
	
	idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < TOTAL_THREADS)
	{
		// The number to test
		num = (START_NUMBER - 1) + (idx * 2);
		for(i = 0; i < n; ++i)
		{
			if(!(num % firstPrimes[i]))
				break;
		}
		if(i == n)
			blockPrimes[threadIdx.x] = num;
		else
			blockPrimes[threadIdx.x] = 0;
	}
	else
	{
		blockPrimes[threadIdx.x] = 0;
	}
	
	__syncthreads();
	
	if(threadIdx.x == 0)
	{
		// sum all the results from the block
		blockSums[blockIdx.x] = 0;
		for(i = 0; i < blockDim.x; ++i)
			blockSums[blockIdx.x] += blockPrimes[i];
	}
}

// main routine that executes on the host
int main(int argc, char *argv[])
{
	//host
	int primes[1024];
	unsigned long long *primeSums;
	int i, j, index;
	int blockSize, nblocks;
	unsigned long long sum;
	size_t len;
	
	//device
	int *primesDevice;
	unsigned long long *primeSumsDevice;
	
	// Find all the primes less than the square root of 2 million ~1414
	primes[0] = 2;
	index = 1;
	sum = 2;
	for(i = 3; i < START_NUMBER; ++i)
	{
		for(j = 0; j < index; ++j)
		{
			if(!(i % primes[j]))
				break;
		}
		if(j == index)
		{
			primes[index++] = i;
			sum += i;
		}
	}
	len = index;
	
	hipMalloc((void **) &primesDevice, len * sizeof(int));
	hipMemcpy(primesDevice, primes, len * sizeof(int), hipMemcpyHostToDevice);
	
	// Test the all odd numbers between 1414 and 2000000
	blockSize = THREADS_PER_BLOCK;
	nblocks = TOTAL_THREADS/blockSize + (TOTAL_THREADS % blockSize?1:0);
	hipMalloc((void **) &primeSumsDevice, nblocks * sizeof(unsigned long long));
	sum_primes <<< nblocks, blockSize >>> (primesDevice, index, primeSumsDevice);
	
	// Retrieve result from device and store it in host array
	primeSums = (unsigned long long *)malloc(nblocks * sizeof(unsigned long long));
	hipMemcpy(primeSums, primeSumsDevice, nblocks * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	for(i = 0; i < nblocks; ++i)
	{
		sum += primeSums[i];
		//printf("%llu\t", primeSums[i]);
	}
	
	// Cleanup
	free(primeSums);
	hipFree(primeSumsDevice);
	hipFree(primesDevice);
	
	// Print results
	printf("%llu\n", sum);
}
