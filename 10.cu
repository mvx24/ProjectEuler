#include "hip/hip_runtime.h"
// nvcc -arch=sm_12 10.cu
// Find the sum of all primes below 2 million

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <sys/types.h>

static struct timeval _stopWatchStartTime, _stopWatchStopTime;

#define InitStopWatch() do{ _stopWatchStartTime.tv_sec = _stopWatchStopTime.tv_sec = 0; _stopWatchStartTime.tv_usec = _stopWatchStopTime.tv_usec = 0; }while(0)
#define StartStopWatch() do{ gettimeofday(&_stopWatchStartTime, NULL); }while(0)
#define StopStopWatch() do{ gettimeofday(&_stopWatchStopTime, NULL); }while(0)
#define GetStopWatchSeconds() ((double)(_stopWatchStopTime.tv_sec - _stopWatchStartTime.tv_sec) + ((double)(_stopWatchStopTime.tv_usec - _stopWatchStartTime.tv_usec)/1000.0))

#define THREADS_PER_BLOCK	512
#define START_NUMBER		1414
#define TOTAL_THREADS		((2000000-START_NUMBER)/2)

// Kernel that executes on the CUDA device
__global__ void sum_primes(int *firstPrimes, size_t n, unsigned long long *blockSums)
{
	__shared__ int blockPrimes[THREADS_PER_BLOCK];
	int i;
	int idx;
	int num;
	
	idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < TOTAL_THREADS)
	{
		// The number to test
		num = (START_NUMBER - 1) + (idx * 2);
		for(i = 0; i < n; ++i)
		{
			if(!(num % firstPrimes[i]))
				break;
		}
		if(i == n)
			blockPrimes[threadIdx.x] = num;
		else
			blockPrimes[threadIdx.x] = 0;
	}
	else
	{
		blockPrimes[threadIdx.x] = 0;
	}
	
	__syncthreads();
	
	if(threadIdx.x == 0)
	{
		// sum all the results from the block
		blockSums[blockIdx.x] = 0;
		for(i = 0; i < blockDim.x; ++i)
			blockSums[blockIdx.x] += blockPrimes[i];
	}
}

// main routine that executes on the host
int main(int argc, char *argv[])
{
	//host
	int primes[1024];
	unsigned long long *primeSums;
	int i, j, index;
	int blockSize, nblocks;
	unsigned long long sum;
	size_t len;
	
	//device
	int *primesDevice;
	unsigned long long *primeSumsDevice;
	
	// Find all the primes less than the square root of 2 million ~1414
	primes[0] = 2;
	index = 1;
	sum = 2;
	for(i = 3; i < START_NUMBER; ++i)
	{
		for(j = 0; j < index; ++j)
		{
			if(!(i % primes[j]))
				break;
		}
		if(j == index)
		{
			primes[index++] = i;
			sum += i;
		}
	}
	len = index;
	
	hipMalloc((void **) &primesDevice, len * sizeof(int));
	hipMemcpy(primesDevice, primes, len * sizeof(int), hipMemcpyHostToDevice);
	
	// Test the all odd numbers between 1414 and 2000000
	blockSize = THREADS_PER_BLOCK;
	nblocks = TOTAL_THREADS/blockSize + (TOTAL_THREADS % blockSize?1:0);
	hipMalloc((void **) &primeSumsDevice, nblocks * sizeof(unsigned long long));
	
	// C++ invocation
	//sum_primes <<< nblocks, blockSize >>> (primesDevice, index, primeSumsDevice);
	// C invocation
	do
	{
		dim3 gridDim;
		dim3 blockDim;
		hipError_t error;
		gridDim.x = nblocks;
		blockDim.x = blockSize;
		gridDim.y = gridDim.z = blockDim.y = blockDim.z = 1;
		error = hipConfigureCall(gridDim, blockDim, 0, NULL);
		if(error != hipSuccess)
		{
			printf("%s\n", hipGetErrorString(error));
			break;
		}
		error = hipSetupArgument(&primesDevice, sizeof(primesDevice), 0);
		error = hipSetupArgument(&index, sizeof(index), sizeof(primesDevice));
		error = hipSetupArgument(&primeSumsDevice, sizeof(primeSumsDevice), sizeof(primesDevice) + sizeof(index));
		printf("Start kernel\n");
		InitStopWatch();
		StartStopWatch();
		error = hipLaunchByPtr(sum_primes);
		if(error != hipSuccess)
		{
			printf("hipLaunchByPtr: %s\n", hipGetErrorString(error));
			break;
		}
		StopStopWatch();
		printf("Kernel completed in %.04f seconds.\n", GetStopWatchSeconds());
	}while(0);
	
	// Retrieve result from device and store it in host array
	primeSums = (unsigned long long *)malloc(nblocks * sizeof(unsigned long long));
	hipMemcpy(primeSums, primeSumsDevice, nblocks * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	for(i = 0; i < nblocks; ++i)
	{
		sum += primeSums[i];
		//printf("%llu\t", primeSums[i]);
	}
	
	// Cleanup
	free(primeSums);
	hipFree(primeSumsDevice);
	hipFree(primesDevice);
	
	// Print results
	printf("%llu\n", sum);
}
